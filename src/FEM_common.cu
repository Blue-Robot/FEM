#include "hip/hip_runtime.h"
#include "FEM_common.h"
#include <stdio.h>
#include "hip/hip_vector_types.h"

const FN_TYPE nMax = 1;
const FN_TYPE D = 0.25;
const FN_TYPE r = 1.52;
const FN_TYPE alpha = 12.02;
const FN_TYPE S = 1;

extern __shared__ FN_TYPE s_mem[];
__global__ void stepKernel(FN_TYPE *nFn_src, FN_TYPE *cFn_src, FN_TYPE *nFn_dst,
		FN_TYPE *cFn_dst, uint *fv, FN_TYPE *fv_weights, uint fv_pitch,
		uint *nbr, FN_TYPE *vtxW, uint vw_pitch, FN_TYPE *vertex_weights, uint vv_pitch, uint vv_size, float4 *grads, uint he_pitch, uint *vertex_parts, uint *block_face_count, double dt) {

	uint size = vertex_parts[blockIdx.x+1] - vertex_parts[blockIdx.x];
	float3 *s_nvGrads = (float3 *)&s_mem[0];
	float3 *s_cvGrads = &s_nvGrads[size];
	FN_TYPE *s_wg = (FN_TYPE *)&s_cvGrads[size];
	for (int i = threadIdx.x; i < 7*size; i += blockDim.x) {
		s_mem[i] = 0.0;
	}

	__syncthreads();

	/* face gradients *************************************/
	if (threadIdx.x >= block_face_count[blockIdx.x])
		return;

	int fn_index[3] = {fv[blockIdx.x*3*fv_pitch + threadIdx.x], fv[blockIdx.x*3*fv_pitch + fv_pitch + threadIdx.x], fv[blockIdx.x*3*fv_pitch + 2*fv_pitch + threadIdx.x]};

	FN_TYPE nv1 = nFn_src[fn_index[2]];
	FN_TYPE nv12 = nFn_src[fn_index[0]] - nv1;
	FN_TYPE nv13 = nFn_src[fn_index[1]] - nv1;
	FN_TYPE cv1 = cFn_src[fn_index[2]];
	FN_TYPE cv12 = cFn_src[fn_index[0]] - cv1;
	FN_TYPE cv13 = cFn_src[fn_index[1]] - cv1;

	fn_index[0] -= vertex_parts[blockIdx.x];
	fn_index[1] -= vertex_parts[blockIdx.x];
	fn_index[2] -= vertex_parts[blockIdx.x];

	float3 grad12 = make_float3(grads[blockIdx.x*2*he_pitch + threadIdx.x]);
	float3 grad13 = make_float3(grads[blockIdx.x*2*he_pitch + he_pitch + threadIdx.x]);

	float3 nvGrad = grad12 * nv12 + grad13 * nv13;
	float3 cvGrad = grad12 * cv12 + grad13 * cv13;


	FN_TYPE weight = fv_weights[blockIdx.x*3*fv_pitch + threadIdx.x];

	for (int i = 0; i < 3; i++) {
		atomicAdd(&s_nvGrads[fn_index[0]].x+i, *(&nvGrad.x+i)*weight);
		atomicAdd(&s_cvGrads[fn_index[0]].x+i, *(&cvGrad.x+i)*weight);
	}
	atomicAdd(&s_wg[fn_index[0]], weight);


	if (fn_index[1] < 0 || fn_index[1] >= size)
		return;

	weight = fv_weights[blockIdx.x*3*fv_pitch + fv_pitch + threadIdx.x];

	atomicAdd(&s_nvGrads[fn_index[1]].x+0, *(&nvGrad.x+0)*weight);
	atomicAdd(&s_nvGrads[fn_index[1]].x+1, *(&nvGrad.x+1)*weight);
	atomicAdd(&s_nvGrads[fn_index[1]].x+2, *(&nvGrad.x+2)*weight);

	atomicAdd(&s_cvGrads[fn_index[1]].x+0, *(&cvGrad.x+0)*weight);
	atomicAdd(&s_cvGrads[fn_index[1]].x+1, *(&cvGrad.x+1)*weight);
	atomicAdd(&s_cvGrads[fn_index[1]].x+2, *(&cvGrad.x+2)*weight);

	atomicAdd(&s_wg[fn_index[1]], weight);

	if (fn_index[2] < 0 || fn_index[2] >= size)
		return;

	weight = fv_weights[blockIdx.x*3*fv_pitch + 2*fv_pitch + threadIdx.x];

	atomicAdd(&s_nvGrads[fn_index[2]].x+0, *(&nvGrad.x+0)*weight);
	atomicAdd(&s_nvGrads[fn_index[2]].x+1, *(&nvGrad.x+1)*weight);
	atomicAdd(&s_nvGrads[fn_index[2]].x+2, *(&nvGrad.x+2)*weight);

	atomicAdd(&s_cvGrads[fn_index[2]].x+0, *(&cvGrad.x+0)*weight);
	atomicAdd(&s_cvGrads[fn_index[2]].x+1, *(&cvGrad.x+1)*weight);
	atomicAdd(&s_cvGrads[fn_index[2]].x+2, *(&cvGrad.x+2)*weight);

	atomicAdd(&s_wg[fn_index[2]], weight);



	// Adjust i
	int i = vertex_parts[blockIdx.x] + threadIdx.x;

	// Kill unnecessary threads
	if (i >= vertex_parts[blockIdx.x + 1])
		return;

	/* laplacian ******************************************/
	FN_TYPE vW = vtxW[blockIdx.x*vw_pitch + threadIdx.x];
	FN_TYPE n = nFn_src[i] * vW;
	FN_TYPE c = cFn_src[i] * vW;

	int end = nbr[blockIdx.x*(vv_size+1)*vv_pitch+threadIdx.x];
	for (int j = 0; j < end; j++) {
		int nIdx = nbr[blockIdx.x*(vv_size+1)*vv_pitch+vv_pitch*(j+1) + threadIdx.x];
		FN_TYPE hW = vertex_weights[blockIdx.x*vv_size*vv_pitch+vv_pitch*j + threadIdx.x];
		n += nFn_src[nIdx] * hW;
		c += cFn_src[nIdx] * hW;
	}

	/* vertex gradients ***********************************/
	__syncthreads();
	FN_TYPE dotP = dot(s_nvGrads[threadIdx.x] / s_wg[threadIdx.x], s_cvGrads[threadIdx.x] / s_wg[threadIdx.x]);
	if (s_wg[threadIdx.x] <= 0) {
		dotP = 0;
	}

	/* update *********************************************/
	FN_TYPE dauN = D * n - alpha * nFn_src[i] * c - alpha * dotP
			+ S * r * nFn_src[i] * (nMax - nFn_src[i]);
	FN_TYPE dauC = c + S * (nFn_src[i] / (1 + nFn_src[i]) - cFn_src[i]);

	nFn_dst[i] = dt * dauN + nFn_src[i];
	cFn_dst[i] = dt * dauC + cFn_src[i];
}

extern "C" void step(FN_TYPE *nFn_src, FN_TYPE *cFn_src, FN_TYPE *nFn_dst,
		FN_TYPE *cFn_dst, uint *fv, FN_TYPE *fv_weights, uint fv_pitchInBytes,
		uint *nbr, FN_TYPE *vtxW, uint vw_pitchInBytes, FN_TYPE *vertex_weights, uint vv_pitchInBytes, uint vv_size, float4 *grads, uint he_pitchInBytes,
		uint *parts_n, uint *block_face_count,
		uint blocks, uint threads, double dt, uint smem_size) {

	dim3 block(threads, 1, 1);
	dim3 grid(blocks, 1, 1);

	stepKernel<<<grid, block, smem_size>>>(nFn_src, cFn_src, nFn_dst, cFn_dst,
			fv, fv_weights, fv_pitchInBytes/sizeof(uint), nbr, vtxW, vw_pitchInBytes/sizeof(uint), vertex_weights, vv_pitchInBytes/sizeof(uint), vv_size, grads, he_pitchInBytes/sizeof(float4), parts_n, block_face_count, dt);

}

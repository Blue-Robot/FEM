#include "hip/hip_runtime.h"
#include "FEM_common.h"
#include <stdio.h>
#include "hip/hip_vector_types.h"

// THRUST
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>

const FN_TYPE nMax = 1;
const FN_TYPE D = 0.25;
const FN_TYPE r = 1.52201704740629;
const FN_TYPE alpha = 12.0228703901698;
const FN_TYPE S = 1;

extern __shared__ FN_TYPE s_mem[];
__global__ void stepKernel(float2 *fn_src, float2 *fn_dst,
		uint *fv, FN_TYPE *fv_weights, uint fv_pitch,
		uint *nbr, FN_TYPE *vtxW, uint vw_pitch, FN_TYPE *vertex_weights, uint vv_pitch, uint vv_size, float4 *grads, uint he_pitch, uint *vertex_parts, uint *block_face_count, double dt) {

	uint size = vertex_parts[blockIdx.x+1] - vertex_parts[blockIdx.x];
	float3 *s_nvGrads = (float3 *)&s_mem[0];
	float3 *s_cvGrads = &s_nvGrads[size];
	FN_TYPE *s_wg = (FN_TYPE *)&s_cvGrads[size];
	for (int i = threadIdx.x; i < 7*size; i += blockDim.x) {
		s_mem[i] = 0.0;
	}

	__syncthreads();

	/* face gradients *************************************/
	if (threadIdx.x >= block_face_count[blockIdx.x])
		return;

	int fn_index[3] = {fv[blockIdx.x*3*fv_pitch + threadIdx.x], fv[blockIdx.x*3*fv_pitch + fv_pitch + threadIdx.x], fv[blockIdx.x*3*fv_pitch + 2*fv_pitch + threadIdx.x]};

	float2 v1 = fn_src[fn_index[2]];
	float2 v12 = fn_src[fn_index[0]] - v1;
	float2 v13 = fn_src[fn_index[1]] - v1;

	fn_index[0] -= vertex_parts[blockIdx.x];
	fn_index[1] -= vertex_parts[blockIdx.x];
	fn_index[2] -= vertex_parts[blockIdx.x];

	float3 grad12 = make_float3(grads[blockIdx.x*2*he_pitch + threadIdx.x]);
	float3 grad13 = make_float3(grads[blockIdx.x*2*he_pitch + he_pitch + threadIdx.x]);

	float3 nvGrad = grad12 * v12.x + grad13 * v13.x;
	float3 cvGrad = grad12 * v12.y + grad13 * v13.y;

	for (int j = 0; j < 3; j++) {
		if (fn_index[j] >= 0 && fn_index[j] < size) {
			FN_TYPE weight = fv_weights[blockIdx.x*3*fv_pitch + j*fv_pitch + threadIdx.x];

			atomicAdd(&s_nvGrads[fn_index[j]].x, nvGrad.x*weight);
			atomicAdd(&s_nvGrads[fn_index[j]].y, nvGrad.y*weight);
			atomicAdd(&s_nvGrads[fn_index[j]].z, nvGrad.z*weight);

			atomicAdd(&s_cvGrads[fn_index[j]].x, cvGrad.x*weight);
			atomicAdd(&s_cvGrads[fn_index[j]].y, cvGrad.y*weight);
			atomicAdd(&s_cvGrads[fn_index[j]].z, cvGrad.z*weight);

			atomicAdd(&s_wg[fn_index[j]], weight);
		}
	}



	// Adjust i
	int i = vertex_parts[blockIdx.x] + threadIdx.x;

	// Kill unnecessary threads
	if (i >= vertex_parts[blockIdx.x + 1])
		return;

	/* laplacian ******************************************/
	double vW = vtxW[blockIdx.x*vw_pitch + threadIdx.x];
	float2 lap = fn_src[i] * vW;

	int end = nbr[blockIdx.x*(vv_size+1)*vv_pitch+threadIdx.x];
	for (int j = 0; j < end; j++) {
		int nIdx = nbr[blockIdx.x*(vv_size+1)*vv_pitch+vv_pitch*(j+1) + threadIdx.x];
		double hW = vertex_weights[blockIdx.x*vv_size*vv_pitch+vv_pitch*j + threadIdx.x];
		lap += fn_src[nIdx] * hW;
	}

	/* vertex gradients ***********************************/
	__syncthreads();
	double dotP = dot(s_nvGrads[threadIdx.x] / s_wg[threadIdx.x], s_cvGrads[threadIdx.x] / s_wg[threadIdx.x]);
	if (s_wg[threadIdx.x] <= 0) {
		dotP = 0;
	}

	/* update *********************************************/
	double dauN = D * lap.x - alpha * fn_src[i].x * lap.y - alpha * dotP
			+ S * r * fn_src[i].x * (nMax - fn_src[i].x);
	double dauC = lap.y + S * (fn_src[i].x / (1 + fn_src[i].x) - fn_src[i].y);

	fn_dst[i].x = dt * dauN + fn_src[i].x > 0 ? dt * dauN + fn_src[i].x : 0.0;
	fn_dst[i].y = dt * dauC + fn_src[i].y > 0 ? dt * dauC + fn_src[i].y : 0.0;
}

extern "C" void step(float2 *fn_src, float2 *fn_dst,
		uint *fv, FN_TYPE *fv_weights, uint fv_pitchInBytes,
		uint *nbr, FN_TYPE *vtxW, uint vw_pitchInBytes, FN_TYPE *vertex_weights, uint vv_pitchInBytes, uint vv_size, float4 *grads, uint he_pitchInBytes,
		uint *parts_n, uint *block_face_count,
		uint blocks, uint threads, double dt, uint smem_size) {

	dim3 block(threads, 1, 1);
	dim3 grid(blocks, 1, 1);

	stepKernel<<<grid, block, smem_size>>>(fn_src, fn_dst,
			fv, fv_weights, fv_pitchInBytes/sizeof(uint), nbr, vtxW, vw_pitchInBytes/sizeof(uint), vertex_weights, vv_pitchInBytes/sizeof(uint), vv_size, grads, he_pitchInBytes/sizeof(float4), parts_n, block_face_count, dt);

}
const float sigma = 0.2;

__global__ void formatKernel(float2 *fn, float4 *vbo, float2 *min, float2 *max, int offset) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	float n = (fn[i].x-(*min).x)/((*max).x-(*min).x);
	n = ((1.0f/(1.0f + expf(-((n-0.5f)/sigma)))) - (1.0f/(1.0f + expf(-((0.0f-0.5f)/sigma)))))/((1.0f/(1.0f + expf(-((1.0f-0.5f)/sigma)))) - (1.0f/(1.0f + expf(-((0.0f-0.5f)/sigma)))));
	//n = n < 0.5 ? pow(n,3.0f) : pow(n,1/3.0f);
	float c = 1-n;
	float b = 1-n;

	vbo[i + offset] = make_float4(n, c, b, 1.0);
//	if (i == 0)
//		printf("%f %f\n", (*max).x, (*min).x);
}

struct comp_x
{
  __host__ __device__
  bool operator()(float2 lhs, float2 rhs)
  {
    return lhs.x < rhs.x;
  }
};


extern "C" void format (float2 *fn, hipGraphicsResource_t *vbo_res, int vertices) {
	thrust::device_ptr<float2> dptr(fn);
	thrust::device_ptr<float2> dresptrmax = thrust::max_element(dptr, dptr + vertices, comp_x());
	thrust::device_ptr<float2> dresptrmin = thrust::min_element(dptr, dptr + vertices, comp_x());

	float2 *max = raw_pointer_cast(dresptrmax);
	float2 *min = raw_pointer_cast(dresptrmin);

	float4 *vboptr;
	size_t num_bytes;

	hipGraphicsMapResources(1, vbo_res, 0);
	hipGraphicsResourceGetMappedPointer((void **)&vboptr, &num_bytes, *vbo_res);

//	float2 *test = new float2[vertices];
//	hipMemcpy(test, fn, vertices*sizeof(float2), hipMemcpyDeviceToHost);
//	for(int i = 0; i < vertices; i++) {
//		printf("%f ", test[i].x);
//	}
//	printf("\n");

	formatKernel<<<vertices,1>>>(fn, vboptr, min, max, vertices);

	hipGraphicsUnmapResources(1, vbo_res, 0);
}

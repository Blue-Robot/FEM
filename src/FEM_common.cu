#include "hip/hip_runtime.h"
#include "FEM_common.h"
#include <stdio.h>
#include "hip/hip_vector_types.h"

const FN_TYPE nMax = 1;
const FN_TYPE D = 0.25;
const FN_TYPE r = 1.52;
const FN_TYPE alpha = 12.02;
const FN_TYPE S = 1;

extern __shared__ FN_TYPE s_mem[];
__global__ void stepKernel(FN_TYPE *nFn_src, FN_TYPE *cFn_src, FN_TYPE *nFn_dst,
		FN_TYPE *cFn_dst, uint *fv, FN_TYPE *fv_weights,
		uint *nbr, FN_TYPE *vtxW, FN_TYPE *vertex_weights, uint vv_pitch, uint vv_size, float3 *grads, uint he_pitch, uint *vertex_parts, uint *face_parts, uint *halo_faces,
		uint *halo_faces_keys, double dt) {

	uint size = vertex_parts[blockIdx.x+1] - vertex_parts[blockIdx.x];
	float3 *s_nvGrads = (float3 *)&s_mem[0];
	float3 *s_cvGrads = &s_nvGrads[size];
	FN_TYPE *s_wg = (FN_TYPE *)&s_cvGrads[size];
	for (int i = threadIdx.x; i < 7*size; i += blockDim.x) {
		s_mem[i] = 0.0;
	}

	__syncthreads();

	/* face gradients *************************************/
	int i = face_parts[blockIdx.x] + threadIdx.x;

	if (i >= face_parts[blockIdx.x + 1]) {
		i = i - face_parts[blockIdx.x + 1] + halo_faces_keys[blockIdx.x];
		if (i >= halo_faces_keys[blockIdx.x + 1])
			return;
		i = halo_faces[i];
	}
	int fn_index[3] = {fv[i * 3], fv[i * 3 + 1], fv[i * 3 + 2]};

	FN_TYPE nv1 = nFn_src[fn_index[2]];
	FN_TYPE nv12 = nFn_src[fn_index[0]] - nv1;
	FN_TYPE nv13 = nFn_src[fn_index[1]] - nv1;
	FN_TYPE cv1 = cFn_src[fn_index[2]];
	FN_TYPE cv12 = cFn_src[fn_index[0]] - cv1;
	FN_TYPE cv13 = cFn_src[fn_index[1]] - cv1;

	fn_index[0] -= vertex_parts[blockIdx.x];
	fn_index[1] -= vertex_parts[blockIdx.x];
	fn_index[2] -= vertex_parts[blockIdx.x];

	float3 grad12 = grads[blockIdx.x*2*he_pitch + threadIdx.x];
	float3 grad13 = grads[blockIdx.x*2*he_pitch + he_pitch + threadIdx.x];

	for (int j = 0; j < 3; j++) {
		if (fn_index[j] >= 0 && fn_index[j] < size) {
			float3 nvGrad = (grad12 * nv12 + grad13 * nv13)*fv_weights[i * 3 + j];
			atomicAdd(&s_nvGrads[fn_index[j]].x, nvGrad.x);
			atomicAdd(&s_nvGrads[fn_index[j]].y, nvGrad.y);
			atomicAdd(&s_nvGrads[fn_index[j]].z, nvGrad.z);

			float3 cvGrad = (grad12 * cv12 + grad13 * cv13)*fv_weights[i * 3 + j];
			atomicAdd(&s_cvGrads[fn_index[j]].x, cvGrad.x);
			atomicAdd(&s_cvGrads[fn_index[j]].y, cvGrad.y);
			atomicAdd(&s_cvGrads[fn_index[j]].z, cvGrad.z);

			atomicAdd(&s_wg[fn_index[j]], fv_weights[i * 3 + j]);
		}
	}

	__syncthreads();

	// Adjust i
	i = vertex_parts[blockIdx.x] + threadIdx.x;

	// Kill unnecessary threads
	if (i >= vertex_parts[blockIdx.x + 1])
		return;

	/* vertex gradients ***********************************/

	FN_TYPE dotP = dot(s_nvGrads[threadIdx.x] / s_wg[threadIdx.x], s_cvGrads[threadIdx.x] / s_wg[threadIdx.x]);
	if (s_wg[threadIdx.x] <= 0)
		dotP = 0;

	/* laplacian ******************************************/
	FN_TYPE vW = vtxW[i];
	FN_TYPE n = nFn_src[i] * vW;
	FN_TYPE c = cFn_src[i] * vW;

	int end = nbr[blockIdx.x*(vv_size+1)*vv_pitch+threadIdx.x];
	for (int j = 0; j < end; j++) {
		int nIdx = nbr[blockIdx.x*(vv_size+1)*vv_pitch+vv_pitch*(j+1) + threadIdx.x];
		FN_TYPE hW = vertex_weights[blockIdx.x*vv_size*vv_pitch+vv_pitch*j + threadIdx.x];
		n += nFn_src[nIdx] * hW;
		c += cFn_src[nIdx] * hW;
	}


	/* update *********************************************/
	FN_TYPE dauN = D * n - alpha * nFn_src[i] * c - alpha * dotP
			+ S * r * nFn_src[i] * (nMax - nFn_src[i]);
	FN_TYPE dauC = c + S * (nFn_src[i] / (1 + nFn_src[i]) - cFn_src[i]);

	nFn_dst[i] = dt * dauN + nFn_src[i];
	cFn_dst[i] = dt * dauC + cFn_src[i];
}

extern "C" void step(FN_TYPE *nFn_src, FN_TYPE *cFn_src, FN_TYPE *nFn_dst,
		FN_TYPE *cFn_dst, uint *fv, FN_TYPE *fv_weights,
		uint *nbr, FN_TYPE *vtxW, FN_TYPE *vertex_weights, uint vv_pitchInBytes, uint vv_size, float3 *grads, uint he_pitchInBytes,
		 uint *parts_n, uint *parts_e, uint *halo_faces,
		uint *halo_faces_keys, uint blocks, uint threads, double dt, uint smem_size) {

	dim3 block(threads, 1, 1);
	dim3 grid(blocks, 1, 1);

	stepKernel<<<grid, block, smem_size>>>(nFn_src, cFn_src, nFn_dst, cFn_dst,
			fv, fv_weights, nbr, vtxW, vertex_weights, vv_pitchInBytes/sizeof(uint), vv_size, grads, he_pitchInBytes/sizeof(float3), parts_n, parts_e, halo_faces, halo_faces_keys, dt);

}

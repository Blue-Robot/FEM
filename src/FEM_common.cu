#include "hip/hip_runtime.h"
#include "FEM_common.h"
#include <stdio.h>
#include "hip/hip_vector_types.h"

const FN_TYPE nMax = 1;
const FN_TYPE D = 0.25;
const FN_TYPE r = 1.52;
const FN_TYPE alpha = 12.02;
const FN_TYPE S = 1;

__global__ void stepKernel(FN_TYPE *nFn_src, FN_TYPE *cFn_src, FN_TYPE *nFn_dst,
		FN_TYPE *cFn_dst, uint *fv, uint *t, uint *nbr, FN_TYPE *vtxW,
		FN_TYPE *heW, float3 *grads, float3 *nfGrads, float3 *cfGrads, uint *vertex_faces,
		FN_TYPE *face_weights, uint vf_pitch, uint *vertex_parts, uint *face_parts,
		uint *halo_faces, uint hf_pitch, double dt) {

	if (threadIdx.x >= halo_faces[blockIdx.x*hf_pitch+1])
				return;

	/* face gradients *************************************/
	int i = face_parts[blockIdx.x] + threadIdx.x;

	if (i >= face_parts[blockIdx.x + 1]) {
		i += halo_faces[blockIdx.x*hf_pitch];

		i = halo_faces[blockIdx.x*hf_pitch + i];
	}

	FN_TYPE nv1 = nFn_src[fv[i * 3 + 2]];
	FN_TYPE nv12 = nFn_src[fv[i * 3]] - nv1;
	FN_TYPE nv13 = nFn_src[fv[i * 3 + 1]] - nv1;
	FN_TYPE cv1 = cFn_src[fv[i * 3 + 2]];
	FN_TYPE cv12 = cFn_src[fv[i * 3]] - cv1;
	FN_TYPE cv13 = cFn_src[fv[i * 3 + 1]] - cv1;

	float3 grad12 = grads[i * 2];
	float3 grad13 = grads[i * 2 + 1];

	nfGrads[i] = grad12 * nv12 + grad13 * nv13;
	cfGrads[i] = grad12 * cv12 + grad13 * cv13;

	__syncthreads();

	// Adjust i
	i = vertex_parts[blockIdx.x] + threadIdx.x;

	// Kill unnecessary threads
	if (i >= vertex_parts[blockIdx.x + 1])
		return;

	/* vertex gradients ***********************************/
	float3 ng = make_float3(0.0f, 0.0f, 0.0f);
	float3 cg = make_float3(0.0f, 0.0f, 0.0f);
	FN_TYPE wg = 0;

	int end = vertex_faces[i];
	for (int j = 0; j < end; j++) {
		uint face = vertex_faces[vf_pitch*(j+1) + i];
		FN_TYPE w = face_weights[vf_pitch*j + i];
		ng += w * nfGrads[face];
		cg += w * cfGrads[face];
		wg += w;
	}
	FN_TYPE dotP = dot(ng, cg)/(wg*wg);
	if (wg <= 0)
		dotP = 0;

	/* laplacian ******************************************/
	FN_TYPE vW = vtxW[i];
	FN_TYPE n = nFn_src[i] * vW;
	FN_TYPE c = cFn_src[i] * vW;

	end = t[i + 1];
	for (int j = t[i]; j < end; j++) {
		int nIdx = nbr[j];
		FN_TYPE hW = heW[j];
		n += nFn_src[nIdx] * hW;
		c += cFn_src[nIdx] * hW;
	}


	/* update *********************************************/
	FN_TYPE dauN = D * n - alpha * nFn_src[i] * c - alpha * dotP
			+ S * r * nFn_src[i] * (nMax - nFn_src[i]);
	FN_TYPE dauC = c + S * (nFn_src[i] / (1 + nFn_src[i]) - cFn_src[i]);

	nFn_dst[i] = dt * dauN + nFn_src[i];
	cFn_dst[i] = dt * dauC + cFn_src[i];
}

extern "C" void step(FN_TYPE *nFn_src, FN_TYPE *cFn_src, FN_TYPE *nFn_dst,
		FN_TYPE *cFn_dst, uint *fv, uint *t, uint *nbr, FN_TYPE *vtxW,
		FN_TYPE *heW, float3 *grads, float3 *nfGrads, float3 *cfGrads, uint *vertex_faces,
		FN_TYPE *face_weights, uint vf_pitchInBytes, uint *parts_n, uint *parts_e,
		uint *halo_faces, uint hf_pitchInBytes, uint blocks, uint threads,
		double dt) {

	dim3 block(threads, 1, 1);
	dim3 grid(blocks, 1, 1);


	stepKernel<<<grid, block>>>(nFn_src, cFn_src, nFn_dst, cFn_dst,
			fv, t, nbr, vtxW, heW, grads, nfGrads, cfGrads, vertex_faces,
			face_weights, vf_pitchInBytes/sizeof(uint), parts_n, parts_e, halo_faces, hf_pitchInBytes/sizeof(uint), dt);

}

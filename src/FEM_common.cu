#include "hip/hip_runtime.h"
#include "FEM_common.h"
#include <stdio.h>
#include "hip/hip_vector_types.h"

const FN_TYPE nMax = 1;
const FN_TYPE D = 0.25;
const FN_TYPE r = 1.52;
const FN_TYPE alpha = 12.02;
const FN_TYPE S = 1;

extern __shared__ FN_TYPE s_mem[];
__global__ void stepKernel(FN_TYPE *nFn_src, FN_TYPE *cFn_src, FN_TYPE *nFn_dst,
		FN_TYPE *cFn_dst, uint *fv, FN_TYPE *fv_weights, uint fv_pitch,
		uint *nbr, FN_TYPE *vtxW, uint vw_pitch, FN_TYPE *vertex_weights, uint vv_pitch, uint vv_size, float4 *grads, uint he_pitch, uint *vertex_parts, uint *halo_vertices, uint hv_pitch, uint* halo_parts, uint *block_face_count, double dt) {

	uint size = vertex_parts[blockIdx.x+1] - vertex_parts[blockIdx.x];
	float3 *s_nvGrads = (float3 *)&s_mem[0];
	float3 *s_cvGrads = &s_nvGrads[size];
	FN_TYPE *s_wg = (FN_TYPE *)&s_cvGrads[size];
	FN_TYPE *s_nFn = &s_wg[size];
	FN_TYPE *s_cFn = &s_nFn[size+halo_parts[blockIdx.x]];
	for (int i = threadIdx.x; i < 7*size; i += blockDim.x) {
		s_mem[i] = 0.0;
	}
	for (int i = threadIdx.x; i < size; i += blockDim.x) {
		s_nFn[i] = nFn_src[vertex_parts[blockIdx.x] + i];
		s_cFn[i] = cFn_src[vertex_parts[blockIdx.x] + i];
	}
	for (int i = threadIdx.x; i < halo_parts[blockIdx.x]; i += blockDim.x) {
		s_nFn[i + size] = nFn_src[halo_vertices[blockIdx.x*hv_pitch + i]];
		s_cFn[i + size] = cFn_src[halo_vertices[blockIdx.x*hv_pitch + i]];
	}


	__syncthreads();

	/* face gradients *************************************/
	if (threadIdx.x >= block_face_count[blockIdx.x])
		return;

	int fn_index[3] = {fv[blockIdx.x*3*fv_pitch + threadIdx.x], fv[blockIdx.x*3*fv_pitch + fv_pitch + threadIdx.x], fv[blockIdx.x*3*fv_pitch + 2*fv_pitch + threadIdx.x]};

	FN_TYPE nv1 = s_nFn[fn_index[2]];
	FN_TYPE nv12 = s_nFn[fn_index[0]] - nv1;
	FN_TYPE nv13 = s_nFn[fn_index[1]] - nv1;
	FN_TYPE cv1 = s_cFn[fn_index[2]];
	FN_TYPE cv12 = s_cFn[fn_index[0]] - cv1;
	FN_TYPE cv13 = s_cFn[fn_index[1]] - cv1;

	float3 grad12 = make_float3(grads[blockIdx.x*2*he_pitch + threadIdx.x]);
	float3 grad13 = make_float3(grads[blockIdx.x*2*he_pitch + he_pitch + threadIdx.x]);

	for (int j = 0; j < 3; j++) {
		if (fn_index[j] >= 0 && fn_index[j] < size) {
			float3 nvGrad = (grad12 * nv12 + grad13 * nv13)*fv_weights[blockIdx.x*3*fv_pitch + j*fv_pitch + threadIdx.x];
			atomicAdd(&s_nvGrads[fn_index[j]].x, nvGrad.x);
			atomicAdd(&s_nvGrads[fn_index[j]].y, nvGrad.y);
			atomicAdd(&s_nvGrads[fn_index[j]].z, nvGrad.z);

			float3 cvGrad = (grad12 * cv12 + grad13 * cv13)*fv_weights[blockIdx.x*3*fv_pitch + j*fv_pitch + threadIdx.x];
			atomicAdd(&s_cvGrads[fn_index[j]].x, cvGrad.x);
			atomicAdd(&s_cvGrads[fn_index[j]].y, cvGrad.y);
			atomicAdd(&s_cvGrads[fn_index[j]].z, cvGrad.z);

			atomicAdd(&s_wg[fn_index[j]], fv_weights[blockIdx.x*3*fv_pitch + j*fv_pitch + threadIdx.x]);
		}
	}

	__syncthreads();

	// Kill unnecessary threads
	if (vertex_parts[blockIdx.x] + threadIdx.x >= vertex_parts[blockIdx.x + 1])
		return;

	/* vertex gradients ***********************************/

	FN_TYPE dotP = dot(s_nvGrads[threadIdx.x] / s_wg[threadIdx.x], s_cvGrads[threadIdx.x] / s_wg[threadIdx.x]);
	if (s_wg[threadIdx.x] <= 0)
		dotP = 0;

	/* laplacian ******************************************/
	FN_TYPE vW = vtxW[blockIdx.x*vw_pitch + threadIdx.x];
	FN_TYPE n = s_nFn[threadIdx.x] * vW;
	FN_TYPE c = s_cFn[threadIdx.x] * vW;

	int end = nbr[blockIdx.x*(vv_size+1)*vv_pitch+threadIdx.x];
	for (int j = 0; j < end; j++) {
		int nIdx = nbr[blockIdx.x*(vv_size+1)*vv_pitch+vv_pitch*(j+1) + threadIdx.x];
		FN_TYPE hW = vertex_weights[blockIdx.x*vv_size*vv_pitch+vv_pitch*j + threadIdx.x];
		n += s_nFn[nIdx] * hW;
		c += s_cFn[nIdx] * hW;
	}


	/* update *********************************************/
	FN_TYPE dauN = D * n - alpha * s_nFn[threadIdx.x] * c - alpha * dotP
			+ S * r * s_nFn[threadIdx.x] * (nMax - s_nFn[threadIdx.x]);
	FN_TYPE dauC = c + S * (s_nFn[threadIdx.x] / (1 + s_nFn[threadIdx.x]) - s_cFn[threadIdx.x]);

	nFn_dst[vertex_parts[blockIdx.x] + threadIdx.x] = dt * dauN + s_nFn[threadIdx.x];
	cFn_dst[vertex_parts[blockIdx.x] + threadIdx.x] = dt * dauC + s_cFn[threadIdx.x];
}

extern "C" void step(FN_TYPE *nFn_src, FN_TYPE *cFn_src, FN_TYPE *nFn_dst,
		FN_TYPE *cFn_dst, uint *fv, FN_TYPE *fv_weights, uint fv_pitchInBytes,
		uint *nbr, FN_TYPE *vtxW, uint vw_pitchInBytes, FN_TYPE *vertex_weights, uint vv_pitchInBytes, uint vv_size, float4 *grads, uint he_pitchInBytes,
		uint *parts_n, uint *halo_vertices, uint hv_pitchInBytes, uint *halo_parts, uint *block_face_count,
		uint blocks, uint threads, double dt, uint smem_size) {

	dim3 block(threads, 1, 1);
	dim3 grid(blocks, 1, 1);

	stepKernel<<<grid, block, smem_size>>>(nFn_src, cFn_src, nFn_dst, cFn_dst,
			fv, fv_weights, fv_pitchInBytes/sizeof(uint), nbr, vtxW, vw_pitchInBytes/sizeof(uint), vertex_weights, vv_pitchInBytes/sizeof(uint), vv_size, grads, he_pitchInBytes/sizeof(float4), parts_n, halo_vertices, hv_pitchInBytes/sizeof(uint), halo_parts, block_face_count, dt);

}

#include "hip/hip_runtime.h"
#include "FEM_common.h"
#include <stdio.h>
#include "hip/hip_vector_types.h"

const FN_TYPE nMax = 1;
const FN_TYPE D = 0.25;
const FN_TYPE r = 1.52;
const FN_TYPE alpha = 12.02;
const FN_TYPE S = 1;

__global__ void stepKernel(FN_TYPE *nFn_src, FN_TYPE *cFn_src, FN_TYPE *nFn_dst,
		FN_TYPE *cFn_dst, uint *fv, FN_TYPE *fv_weights, uint *t,
		uint *nbr, FN_TYPE *vtxW, FN_TYPE *heW, float3 *grads, float3 *nvGrads,
		float3 *cvGrads, FN_TYPE *wg, uint *f, uint *faces,
		FN_TYPE *fW, uint *vertex_parts, uint *face_parts, uint *halo_faces,
		uint *halo_faces_keys, double dt) {

	/* face gradients *************************************/
	int i = face_parts[blockIdx.x] + threadIdx.x;

	if (i >= face_parts[blockIdx.x + 1]) {

		i = i - face_parts[blockIdx.x + 1] + halo_faces_keys[blockIdx.x];
		if (i >= halo_faces_keys[blockIdx.x + 1])
			return;
		i = halo_faces[i];
	}
	int first = fv[i * 3];
	int second = fv[i * 3 + 1];
	int third = fv[i * 3 + 2];

	FN_TYPE nv1 = nFn_src[third];
	FN_TYPE nv12 = nFn_src[first] - nv1;
	FN_TYPE nv13 = nFn_src[second] - nv1;
	FN_TYPE cv1 = cFn_src[third];
	FN_TYPE cv12 = cFn_src[first] - cv1;
	FN_TYPE cv13 = cFn_src[second] - cv1;

	float3 grad12 = grads[i * 2];
	float3 grad13 = grads[i * 2 + 1];

	float3 nvGrad;
	float3 cvGrad;
	if (first >= vertex_parts[blockIdx.x] && first < vertex_parts[blockIdx.x+1]) {
	nvGrad = (grad12 * nv12 + grad13 * nv13)*fv_weights[i * 3];
	atomicAdd(&nvGrads[first].x, nvGrad.x);
	atomicAdd(&nvGrads[first].y, nvGrad.y);
	atomicAdd(&nvGrads[first].z, nvGrad.z);


	cvGrad = (grad12 * cv12 + grad13 * cv13)*fv_weights[i * 3];
	atomicAdd(&cvGrads[first].x, cvGrad.x);
	atomicAdd(&cvGrads[first].y, cvGrad.y);
	atomicAdd(&cvGrads[first].z, cvGrad.z);

	atomicAdd(&wg[first], fv_weights[i * 3]);
	}

	if (second >= vertex_parts[blockIdx.x] && second < vertex_parts[blockIdx.x+1]) {
	nvGrad = (grad12 * nv12 + grad13 * nv13)*fv_weights[i * 3 + 1];
	atomicAdd(&nvGrads[second].x, nvGrad.x);
	atomicAdd(&nvGrads[second].y, nvGrad.y);
	atomicAdd(&nvGrads[second].z, nvGrad.z);

	cvGrad = (grad12 * cv12 + grad13 * cv13)*fv_weights[i * 3 + 1];
	atomicAdd(&cvGrads[second].x, cvGrad.x);
	atomicAdd(&cvGrads[second].y, cvGrad.y);
	atomicAdd(&cvGrads[second].z, cvGrad.z);

	atomicAdd(&wg[second], fv_weights[i * 3 + 1]);
	}


	if (third >= vertex_parts[blockIdx.x] && third < vertex_parts[blockIdx.x+1]) {
	nvGrad = (grad12 * nv12 + grad13 * nv13)*fv_weights[i * 3 + 2];
	atomicAdd(&nvGrads[third].x, nvGrad.x);
	atomicAdd(&nvGrads[third].y, nvGrad.y);
	atomicAdd(&nvGrads[third].z, nvGrad.z);

	cvGrad = (grad12 * cv12 + grad13 * cv13)*fv_weights[i * 3 + 2];
	atomicAdd(&cvGrads[third].x, cvGrad.x);
	atomicAdd(&cvGrads[third].y, cvGrad.y);
	atomicAdd(&cvGrads[third].z, cvGrad.z);

	atomicAdd(&wg[third], fv_weights[i * 3 + 2]);
	}


	__syncthreads();

	// Adjust i
	i = vertex_parts[blockIdx.x] + threadIdx.x;

	// Kill unnecessary threads
	if (i >= vertex_parts[blockIdx.x + 1])
		return;

	/* vertex gradients ***********************************/

	FN_TYPE dotP = dot(nvGrads[i] / wg[i], cvGrads[i] / wg[i]);
	if (wg[i] <= 0)
		dotP = 0;

	/* laplacian ******************************************/
	FN_TYPE vW = vtxW[i];
	FN_TYPE n = nFn_src[i] * vW;
	FN_TYPE c = cFn_src[i] * vW;

	int end = t[i + 1];
	for (int j = t[i]; j < end; j++) {
		int nIdx = nbr[j];
		FN_TYPE hW = heW[j];
		n += nFn_src[nIdx] * hW;
		c += cFn_src[nIdx] * hW;
	}


	/* update *********************************************/
	FN_TYPE dauN = D * n - alpha * nFn_src[i] * c - alpha * dotP
			+ S * r * nFn_src[i] * (nMax - nFn_src[i]);
	FN_TYPE dauC = c + S * (nFn_src[i] / (1 + nFn_src[i]) - cFn_src[i]);

	nFn_dst[i] = dt * dauN + nFn_src[i];
	cFn_dst[i] = dt * dauC + cFn_src[i];
}

extern "C" void step(FN_TYPE *nFn_src, FN_TYPE *cFn_src, FN_TYPE *nFn_dst,
		FN_TYPE *cFn_dst, uint *fv, FN_TYPE *fv_weights, uint *t,
		uint *nbr, FN_TYPE *vtxW, FN_TYPE *heW, float3 *grads, float3 *nvGrads,
		float3 *cvGrads, FN_TYPE *wg, uint *f, uint *faces,
		FN_TYPE *fW, uint *parts_n, uint *parts_e, uint *halo_faces,
		uint *halo_faces_keys, uint blocks, uint threads, double dt) {

	dim3 block(threads, 1, 1);
	dim3 grid(blocks, 1, 1);

	stepKernel<<<grid, block>>>(nFn_src, cFn_src, nFn_dst, cFn_dst,
			fv, fv_weights, t, nbr, vtxW, heW, grads, nvGrads, cvGrads, wg, f,
			faces, fW, parts_n, parts_e, halo_faces, halo_faces_keys, dt);

}

#include "hip/hip_runtime.h"
#include "FEM_common.h"
#include <stdio.h>
#include "hip/hip_vector_types.h"

const FN_TYPE nMax = 1;
const FN_TYPE D = 0.25;
const FN_TYPE r = 1.52;
const FN_TYPE alpha = 12.02;
const FN_TYPE S = 1;

__global__ void stepKernel(FN_TYPE *nFn_src, FN_TYPE *cFn_src, FN_TYPE *nFn_dst,
		FN_TYPE *cFn_dst, uint *fv, uint fv_pitch, uint *nbr, FN_TYPE *vertex_weights, uint vv_pitch, FN_TYPE *vtxW,
		float3 *grads, float3 *nfGrads, float3 *cfGrads, uint *vertex_faces,
		FN_TYPE *face_weights, uint vf_pitch, uint *vertex_parts, uint *face_parts,
		uint *halo_faces, uint hf_pitch, double dt) {

	if (threadIdx.x >= halo_faces[blockIdx.x*hf_pitch+1])
				return;

	/* face gradients *************************************/
	int i = face_parts[blockIdx.x] + threadIdx.x;

	if (i >= face_parts[blockIdx.x + 1]) {
		i += halo_faces[blockIdx.x*hf_pitch];

		i = halo_faces[blockIdx.x*hf_pitch + i];
	}

	FN_TYPE nv1 = nFn_src[fv[2*fv_pitch + i]];
	FN_TYPE nv12 = nFn_src[fv[i]] - nv1;
	FN_TYPE nv13 = nFn_src[fv[fv_pitch + i]] - nv1;
	FN_TYPE cv1 = cFn_src[fv[2*fv_pitch + i]];
	FN_TYPE cv12 = cFn_src[fv[i]] - cv1;
	FN_TYPE cv13 = cFn_src[fv[fv_pitch + i]] - cv1;

	float3 grad12 = grads[i * 2];
	float3 grad13 = grads[i * 2 + 1];

	nfGrads[i] = grad12 * nv12 + grad13 * nv13;
	cfGrads[i] = grad12 * cv12 + grad13 * cv13;

	__syncthreads();

	// Adjust i
	i = vertex_parts[blockIdx.x] + threadIdx.x;

	// Kill unnecessary threads
	if (i >= vertex_parts[blockIdx.x + 1])
		return;

	/* vertex gradients ***********************************/
	float3 ng = make_float3(0.0f, 0.0f, 0.0f);
	float3 cg = make_float3(0.0f, 0.0f, 0.0f);
	FN_TYPE wg = 0;

	int end = vertex_faces[i];
	for (int j = 0; j < end; j++) {
		uint face = vertex_faces[vf_pitch*(j+1) + i];
		FN_TYPE w = face_weights[vf_pitch*j + i];
		ng += w * nfGrads[face];
		cg += w * cfGrads[face];
		wg += w;
	}
	FN_TYPE dotP = dot(ng, cg)/(wg*wg);
	if (wg <= 0)
		dotP = 0;

	/* laplacian ******************************************/
	FN_TYPE vW = vtxW[i];
	FN_TYPE n = nFn_src[i] * vW;
	FN_TYPE c = cFn_src[i] * vW;

	end = nbr[i];
	for (int j = 0; j < end; j++) {
		int nIdx = nbr[vv_pitch*(j+1) + i];
		FN_TYPE hW = vertex_weights[vv_pitch*j + i];
		n += nFn_src[nIdx] * hW;
		c += cFn_src[nIdx] * hW;
	}


	/* update *********************************************/
	FN_TYPE dauN = D * n - alpha * nFn_src[i] * c - alpha * dotP
			+ S * r * nFn_src[i] * (nMax - nFn_src[i]);
	FN_TYPE dauC = c + S * (nFn_src[i] / (1 + nFn_src[i]) - cFn_src[i]);

	nFn_dst[i] = dt * dauN + nFn_src[i];
	cFn_dst[i] = dt * dauC + cFn_src[i];
}

extern "C" void step(FN_TYPE *nFn_src, FN_TYPE *cFn_src, FN_TYPE *nFn_dst,
		FN_TYPE *cFn_dst, uint *face_vertices, uint fv_pitchInBytes, uint *nbr, FN_TYPE *vertex_weights, uint vv_pitchInBytes,
		FN_TYPE *vtxW, float3 *grads, float3 *nfGrads, float3 *cfGrads, uint *vertex_faces,
		FN_TYPE *face_weights, uint vf_pitchInBytes, uint *parts_n, uint *parts_e,
		uint *halo_faces, uint hf_pitchInBytes, uint blocks, uint threads,
		double dt) {

	dim3 block(threads, 1, 1);
	dim3 grid(blocks, 1, 1);


	stepKernel<<<grid, block>>>(nFn_src, cFn_src, nFn_dst, cFn_dst,
			face_vertices, fv_pitchInBytes/sizeof(uint), nbr, vertex_weights, vv_pitchInBytes/sizeof(uint), vtxW, grads, nfGrads, cfGrads, vertex_faces,
			face_weights, vf_pitchInBytes/sizeof(uint), parts_n, parts_e, halo_faces, hf_pitchInBytes/sizeof(uint), dt);

}

#include "hip/hip_runtime.h"
#include "FEM_common.h"
#include <stdio.h>
#include "hip/hip_vector_types.h"

const FN_TYPE nMax = 1;
const FN_TYPE D = 0.25;
const FN_TYPE r = 1.52;
const FN_TYPE alpha = 12.02;
const FN_TYPE S = 1;

__global__ void stepKernel(FN_TYPE *nFn_src, FN_TYPE *cFn_src, FN_TYPE *nFn_dst,
		FN_TYPE *cFn_dst, FN_TYPE *nLap, FN_TYPE *cLap, uint *fv, uint *t,
		uint *nbr, FN_TYPE *vtxW, FN_TYPE *heW, float3 *grads, float3 *nfGrads,
		float3 *cfGrads, float3 *nvGrads, float3 *cvGrads, uint *f, uint *faces,
		FN_TYPE *fW, uint *vertex_parts, uint *face_parts, uint *halo_faces,
		uint *halo_faces_keys, double dt) {

	/* face gradients *************************************/
	int i = face_parts[blockIdx.x] + threadIdx.x;

	if (i >= face_parts[blockIdx.x + 1]) {

		i = i - face_parts[blockIdx.x + 1] + halo_faces_keys[blockIdx.x];
		if (i >= halo_faces_keys[blockIdx.x + 1])
			return;
		i = halo_faces[i];
	}

	FN_TYPE nv1 = nFn_src[fv[i * 3 + 2]];
	FN_TYPE nv12 = nFn_src[fv[i * 3]] - nv1;
	FN_TYPE nv13 = nFn_src[fv[i * 3 + 1]] - nv1;
	FN_TYPE cv1 = cFn_src[fv[i * 3 + 2]];
	FN_TYPE cv12 = cFn_src[fv[i * 3]] - cv1;
	FN_TYPE cv13 = cFn_src[fv[i * 3 + 1]] - cv1;

	float3 grad12 = grads[i * 2];
	float3 grad13 = grads[i * 2 + 1];

	nfGrads[i] = grad12 * nv12 + grad13 * nv13;
	cfGrads[i] = grad12 * cv12 + grad13 * cv13;

	__syncthreads();

	// Adjust i
	i = vertex_parts[blockIdx.x] + threadIdx.x;

	// Kill unnecessary threads
	if (i >= vertex_parts[blockIdx.x + 1])
		return;

	/* vertex gradients ***********************************/
	float3 ng = make_float3(0.0f, 0.0f, 0.0f);
	float3 cg = make_float3(0.0f, 0.0f, 0.0f);
	FN_TYPE wg = 0;

	int end = f[i + 1];

	for (int j = f[i]; j < end; j++) {
		uint face = faces[j];
		FN_TYPE w = fW[j];
		ng += w * nfGrads[face];
		cg += w * cfGrads[face];
		wg += w;
	}
	if (wg > 0) {
		nvGrads[i] = ng / wg;
		cvGrads[i] = cg / wg;
	} else {
		nvGrads[i] = make_float3(0.0f, 0.0f, 0.0f);
		cvGrads[i] = make_float3(0.0f, 0.0f, 0.0f);
	}

	/* laplacian ******************************************/
	FN_TYPE vW = vtxW[i];
	FN_TYPE n = nFn_src[i] * vW;
	FN_TYPE c = cFn_src[i] * vW;

	end = t[i + 1];
	for (int j = t[i]; j < end; j++) {
		int nIdx = nbr[j];
		FN_TYPE hW = heW[j];
		n += nFn_src[nIdx] * hW;
		c += cFn_src[nIdx] * hW;
	}
	nLap[i] = n;
	cLap[i] = c;


	/* update *********************************************/
	float3 nVG = nvGrads[i];
	float3 cVG = cvGrads[i];
	FN_TYPE dotP = dot(nVG, cVG);

	FN_TYPE dauN = D * nLap[i] - alpha * nFn_src[i] * cLap[i] - alpha * dotP
			+ S * r * nFn_src[i] * (nMax - nFn_src[i]);
	FN_TYPE dauC = cLap[i] + S * (nFn_src[i] / (1 + nFn_src[i]) - cFn_src[i]);

	nFn_dst[i] = dt * dauN + nFn_src[i];
	cFn_dst[i] = dt * dauC + cFn_src[i];
}

extern "C" void step(FN_TYPE *nFn_src, FN_TYPE *cFn_src, FN_TYPE *nFn_dst,
		FN_TYPE *cFn_dst, FN_TYPE *nLap, FN_TYPE *cLap, uint *fv, uint *t,
		uint *nbr, FN_TYPE *vtxW, FN_TYPE *heW, float3 *grads, float3 *nfGrads,
		float3 *cfGrads, float3 *nvGrads, float3 *cvGrads, uint *f, uint *faces,
		FN_TYPE *fW, uint *parts_n, uint *parts_e, uint *halo_faces,
		uint *halo_faces_keys, uint blocks, uint threads, double dt) {

	dim3 block(threads, 1, 1);
	dim3 grid(blocks, 1, 1);

	stepKernel<<<grid, block>>>(nFn_src, cFn_src, nFn_dst, cFn_dst, nLap, cLap,
			fv, t, nbr, vtxW, heW, grads, nfGrads, cfGrads, nvGrads, cvGrads, f,
			faces, fW, parts_n, parts_e, halo_faces, halo_faces_keys, dt);

}

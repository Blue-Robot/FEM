#include "hip/hip_runtime.h"
#include "FEM_common.h"
#include <stdio.h>
#include "hip/hip_vector_types.h"
#include <hip/hip_runtime_api.h>

const FN_TYPE nMax = 1;
const FN_TYPE D = 0.25;
const FN_TYPE r = 1.52;
const FN_TYPE alpha = 12.02;
const FN_TYPE S = 1;

texture<float, 1, hipReadModeElementType> nTex1;
texture<float, 1, hipReadModeElementType> cTex1;
texture<float, 1, hipReadModeElementType> nTex2;
texture<float, 1, hipReadModeElementType> cTex2;

extern __shared__ FN_TYPE s_mem[];
__global__ void stepKernel(FN_TYPE *nFn_src, FN_TYPE *cFn_src, FN_TYPE *nFn_dst,
		FN_TYPE *cFn_dst, uint *fv, FN_TYPE *fv_weights, uint fv_pitch,
		uint *nbr, FN_TYPE *vtxW, uint vw_pitch, FN_TYPE *vertex_weights, uint vv_pitch, uint vv_size, float4 *grads, uint he_pitch, uint *vertex_parts, uint *block_face_count, double dt, bool one) {

	uint size = vertex_parts[blockIdx.x+1] - vertex_parts[blockIdx.x];
	float3 *s_nvGrads = (float3 *)&s_mem[0];
	float3 *s_cvGrads = &s_nvGrads[size];
	FN_TYPE *s_wg = (FN_TYPE *)&s_cvGrads[size];
	for (int i = threadIdx.x; i < 7*size; i += blockDim.x) {
		s_mem[i] = 0.0;
	}

	__syncthreads();

	/* face gradients *************************************/
	if (threadIdx.x >= block_face_count[blockIdx.x])
		return;

	int fn_index[3] = {fv[blockIdx.x*3*fv_pitch + threadIdx.x], fv[blockIdx.x*3*fv_pitch + fv_pitch + threadIdx.x], fv[blockIdx.x*3*fv_pitch + 2*fv_pitch + threadIdx.x]};

	FN_TYPE nv1 = one ? tex1Dfetch(nTex1, fn_index[2]) : tex1Dfetch(nTex2, fn_index[2]);
	FN_TYPE nv12 = (one ? tex1Dfetch(nTex1, fn_index[0]) : tex1Dfetch(nTex2, fn_index[0])) - nv1;
	FN_TYPE nv13 = (one ? tex1Dfetch(nTex1, fn_index[1]) : tex1Dfetch(nTex2, fn_index[1])) - nv1;
	FN_TYPE cv1 = one ? tex1Dfetch(cTex1, fn_index[2]) : tex1Dfetch(cTex2, fn_index[2]);
	FN_TYPE cv12 = (one ? tex1Dfetch(cTex1, fn_index[0]) : tex1Dfetch(cTex2, fn_index[0])) - cv1;
	FN_TYPE cv13 = (one ? tex1Dfetch(cTex1, fn_index[1]) : tex1Dfetch(cTex2, fn_index[1])) - cv1;

	fn_index[0] -= vertex_parts[blockIdx.x];
	fn_index[1] -= vertex_parts[blockIdx.x];
	fn_index[2] -= vertex_parts[blockIdx.x];

	float3 grad12 = make_float3(grads[blockIdx.x*2*he_pitch + threadIdx.x]);
	float3 grad13 = make_float3(grads[blockIdx.x*2*he_pitch + he_pitch + threadIdx.x]);

	for (int j = 0; j < 3; j++) {
		if (fn_index[j] >= 0 && fn_index[j] < size) {
			float3 nvGrad = (grad12 * nv12 + grad13 * nv13)*fv_weights[blockIdx.x*3*fv_pitch + j*fv_pitch + threadIdx.x];
			atomicAdd(&s_nvGrads[fn_index[j]].x, nvGrad.x);
			atomicAdd(&s_nvGrads[fn_index[j]].y, nvGrad.y);
			atomicAdd(&s_nvGrads[fn_index[j]].z, nvGrad.z);

			float3 cvGrad = (grad12 * cv12 + grad13 * cv13)*fv_weights[blockIdx.x*3*fv_pitch + j*fv_pitch + threadIdx.x];
			atomicAdd(&s_cvGrads[fn_index[j]].x, cvGrad.x);
			atomicAdd(&s_cvGrads[fn_index[j]].y, cvGrad.y);
			atomicAdd(&s_cvGrads[fn_index[j]].z, cvGrad.z);

			atomicAdd(&s_wg[fn_index[j]], fv_weights[blockIdx.x*3*fv_pitch + j*fv_pitch + threadIdx.x]);
		}
	}

	__syncthreads();

	// Adjust i
	int i = vertex_parts[blockIdx.x] + threadIdx.x;

	// Kill unnecessary threads
	if (i >= vertex_parts[blockIdx.x + 1])
		return;

	/* vertex gradients ***********************************/

	FN_TYPE dotP = dot(s_nvGrads[threadIdx.x] / s_wg[threadIdx.x], s_cvGrads[threadIdx.x] / s_wg[threadIdx.x]);
	if (s_wg[threadIdx.x] <= 0)
		dotP = 0;

	/* laplacian ******************************************/
	FN_TYPE vW = vtxW[blockIdx.x*vw_pitch + threadIdx.x];
	FN_TYPE n = nFn_src[i] * vW;
	FN_TYPE c = cFn_src[i] * vW;

	int end = nbr[blockIdx.x*(vv_size+1)*vv_pitch+threadIdx.x];
	for (int j = 0; j < end; j++) {
		int nIdx = nbr[blockIdx.x*(vv_size+1)*vv_pitch+vv_pitch*(j+1) + threadIdx.x];
		FN_TYPE hW = vertex_weights[blockIdx.x*vv_size*vv_pitch+vv_pitch*j + threadIdx.x];
		n += (one ? tex1Dfetch(nTex1, nIdx) : tex1Dfetch(nTex2, nIdx)) * hW;
		c += (one ? tex1Dfetch(cTex1, nIdx) : tex1Dfetch(cTex2, nIdx)) * hW;
	}


	/* update *********************************************/
	FN_TYPE dauN = D * n - alpha * nFn_src[i] * c - alpha * dotP
			+ S * r * nFn_src[i] * (nMax - nFn_src[i]);
	FN_TYPE dauC = c + S * (nFn_src[i] / (1 + nFn_src[i]) - cFn_src[i]);

	nFn_dst[i] = dt * dauN + nFn_src[i];
	cFn_dst[i] = dt * dauC + cFn_src[i];
}

extern "C" void step(FN_TYPE *nFn_src, FN_TYPE *cFn_src, FN_TYPE *nFn_dst,
		FN_TYPE *cFn_dst, uint *fv, FN_TYPE *fv_weights, uint fv_pitchInBytes,
		uint *nbr, FN_TYPE *vtxW, uint vw_pitchInBytes, FN_TYPE *vertex_weights, uint vv_pitchInBytes, uint vv_size, float4 *grads, uint he_pitchInBytes,
		uint *parts_n, uint *block_face_count,
		uint blocks, uint threads, double dt, uint smem_size, bool one) {

	dim3 block(threads, 1, 1);
	dim3 grid(blocks, 1, 1);

	stepKernel<<<grid, block, smem_size>>>(nFn_src, cFn_src, nFn_dst, cFn_dst,
			fv, fv_weights, fv_pitchInBytes/sizeof(uint), nbr, vtxW, vw_pitchInBytes/sizeof(uint), vertex_weights, vv_pitchInBytes/sizeof(uint), vv_size, grads, he_pitchInBytes/sizeof(float4), parts_n, block_face_count, dt, one);
}

extern "C" void bindN1Texture(FN_TYPE *cuArray, hipChannelFormatDesc channelDesc, size_t size) {
nTex1.normalized = false;
nTex1.filterMode = hipFilterModePoint;
nTex1.addressMode[0] = hipAddressModeClamp;

checkCudaErrors(hipBindTexture(0, nTex1, cuArray, channelDesc, size));
}

extern "C" void bindC1Texture(FN_TYPE *cuArray, hipChannelFormatDesc channelDesc, size_t size) {
cTex1.normalized = false;
cTex1.filterMode = hipFilterModePoint;
cTex1.addressMode[0] = hipAddressModeClamp;

checkCudaErrors(hipBindTexture(0, cTex1, cuArray, channelDesc, size));
}

extern "C" void bindN2Texture(FN_TYPE *cuArray, hipChannelFormatDesc channelDesc, size_t size) {
nTex2.normalized = false;
nTex2.filterMode = hipFilterModePoint;
nTex2.addressMode[0] = hipAddressModeClamp;

checkCudaErrors(hipBindTexture(0, nTex2, cuArray, channelDesc, size));
}

extern "C" void bindC2Texture(FN_TYPE *cuArray, hipChannelFormatDesc channelDesc, size_t size) {
cTex2.normalized = false;
cTex2.filterMode = hipFilterModePoint;
cTex2.addressMode[0] = hipAddressModeClamp;

checkCudaErrors(hipBindTexture(0, cTex2, cuArray, channelDesc, size));
}

extern "C" void unbindN1Texture() {
checkCudaErrors(hipUnbindTexture(nTex1));
}

extern "C" void unbindC1Texture() {
checkCudaErrors(hipUnbindTexture(cTex1));
}

extern "C" void unbindN2Texture() {
checkCudaErrors(hipUnbindTexture(nTex2));
}

extern "C" void unbindC2Texture() {
checkCudaErrors(hipUnbindTexture(cTex2));
}
